#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <cstdio>
#include <stdlib.h>

#include "board.h"
#include "board_factory.h"
#include "board.cpp"

extern "C" {



__device__ int evaluate(Board board,unsigned int*random_numbers,unsigned int numberOfThreads,unsigned int globalId)
{
	int resultMultiplier=1;
	unsigned int correct_moves[64];
	for(unsigned int iteration=0;!board.is_game_ended();iteration++)
	{
		if(!board.can_player_put_piece())
		{
			board=board.pass_turn();
		}
		else
		{
			unsigned int idx=0;
			for(int i=0;i<64;i++)
				if(board.is_correct_move(i))
					correct_moves[idx++]=i;
			unsigned int element_number=random_numbers[numberOfThreads*iteration+globalId]%idx;
			board=board.make_move(correct_moves[element_number]);
		}
		resultMultiplier*=-1;

	}
	switch(board.get_dominating_site())
	{
		case Board::PLAYER:
			return 1*resultMultiplier;
		case Board::OPPONENT:
			return -1*resultMultiplier;
		case Board::NONE:
			return 0*resultMultiplier;
	}
	return -1;
}

__global__ void check_move(ull player_pieces,ull opponent_pieces,int*moves_to_check,unsigned int*randoms,int*result)
{
	int moveId=blockIdx.y;
	int thidX=(blockIdx.x*blockDim.x)+threadIdx.x;
	int numberOfTries=blockDim.x*gridDim.x;
	int threadGlobalId=numberOfTries*blockIdx.y+thidX;
	Board computedBoard(player_pieces,opponent_pieces);
	int checkedMove=moves_to_check[moveId];
	Board movedBoard=computedBoard.make_move(checkedMove);
	result[threadGlobalId]=-evaluate(movedBoard,randoms,numberOfTries,threadGlobalId);
}
}


